#include "hip/hip_runtime.h"
﻿#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>

#define THREAD_PER_BLOCK 256

// bank conflict
__global__ void reduce1(float* d_in, float* d_out) {
	__shared__ float sdata[THREAD_PER_BLOCK];

	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = d_in[i];
	__syncthreads();

	// do reduction in shared mem
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		int index = 2 * s * tid;
		if (index < blockDim.x) {
			sdata[index] += sdata[index + s];
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if (tid == 0) d_out[blockIdx.x] = sdata[0];
}

bool check(float* out, float* res, int n) {
	for (int i = 0; i < n; i++) {
		if (out[i] != res[i])
			return false;
	}
	return true;
}

int main() {
	const int N = 32 * 1024 * 1024;
	float* a = (float*)malloc(N * sizeof(float));
	float* d_a;
	hipMalloc((void**)&d_a, N * sizeof(float));

	int block_num = N / THREAD_PER_BLOCK;
	float* out = (float*)malloc((N / THREAD_PER_BLOCK) * sizeof(float));
	float* d_out;
	hipMalloc((void**)&d_out, (N / THREAD_PER_BLOCK) * sizeof(float));
	float* res = (float*)malloc((N / THREAD_PER_BLOCK) * sizeof(float));

	for (int i = 0; i < N; i++) {
		a[i] = 1;
	}

	for (int i = 0; i < block_num; i++) {
		float cur = 0;
		for (int j = 0; j < THREAD_PER_BLOCK; j++) {
			cur += a[i * THREAD_PER_BLOCK + j];
		}
		res[i] = cur;
	}

	hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

	dim3 Grid(N / THREAD_PER_BLOCK, 1);
	dim3 Block(THREAD_PER_BLOCK, 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float msecTotal = 0;
	int nIter = 1000;
	hipEventRecord(start);
	for (int i = 0; i < nIter; i++) {
		reduce1 << <Grid, Block >> > (d_a, d_out);

	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&msecTotal, start, stop);

	hipMemcpy(out, d_out, block_num * sizeof(float), hipMemcpyDeviceToHost);

	if (check(out, res, block_num))printf("the ans is right, time : %f\n", msecTotal);
	else {
		printf("the ans is wrong\n");
		for (int i = 0; i < block_num; i++) {
			printf("%lf ", out[i]);
		}
		printf("\n");
	}

	hipFree(d_a);
	hipFree(d_out);
}
﻿#include "reduce6.cuh"
#include<iostream>
#include <malloc.h>

using namespace std;


bool check(float* out, float* res, int n) {
    for (int i = 0; i < n; i++) {
        if (out[i] != res[i])
            return false;
    }
    return true;
}

int main()
{
    int block_num = N / Num_per_block;
    long int size = N * sizeof(float);
    float* in = (float*)malloc(size);
    float* res = (float*)malloc(block_num * sizeof(float));
    float* d_in;
    float* d_out;

    for (long int i = 0; i < N; i++) {
        in[i] = 1.0;
    }
    for (int i = 0; i < block_num; i++) {
        float cur = 0;
        for (int j = 0; j < Num_per_block; j++) {
            cur += in[i * Num_per_block + j];
        }
        res[i] = cur;
    }

    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out,block_num*sizeof(float));
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

    dim3 gridsize(block_num);
    dim3 blocksize(Thread_per_block);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    for (int i = 0; i < 1000; i++) {
        reduce6<Thread_per_block><<<gridsize, blocksize >>> (d_in, d_out,N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float   elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    float* out = (float*)malloc(block_num * sizeof(float));
    hipMemcpy(out, d_out, N / Num_per_block * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  
    if (check(out, res, block_num))printf("the ans is right\n");
    else {
        printf("the ans is wrong\n");
        for (int i = 0; i < block_num; i++) {
            printf("%lf ", out[i]);
        }
        printf("\n");
    }
    cout << "Time is " << elapsedTime <<" ms " << endl;
    hipFree(d_in);
    hipFree(d_out);
    free(in);


    return 0;
}


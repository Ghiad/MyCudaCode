#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<iostream>
#include <malloc.h>
using namespace std;

const long int N = 32 * 1024 * 1024;

const int Thread_per_block = 256;

bool check(float* out, float* res, int n) {
    for (int i = 0; i < n; i++) {
        if (out[i] != res[i])
            return false;
    }
    return true;
}
template <unsigned int blockSize>
__device__ void warpReduce(volatile float* cache, int tid) {
    if (blockSize >= 64)cache[tid] += cache[tid + 32];
    if (blockSize >= 32)cache[tid] += cache[tid + 16];
    if (blockSize >= 16)cache[tid] += cache[tid + 8];
    if (blockSize >= 8)cache[tid] += cache[tid + 4];
    if (blockSize >= 4)cache[tid] += cache[tid + 2];
    if (blockSize >= 2)cache[tid] += cache[tid + 1];
}
template <unsigned int blockSize>
__global__ void reduce6(float* d_in, float* d_out, int n)
{
    __shared__ float sdata[blockSize];
    unsigned int tid = threadIdx.x;
    unsigned int id = (2 * blockSize) * blockIdx.x + tid;
    unsigned int gridSize = 2 * blockSize * gridDim.x;
    //sdata[tid] = 0.0;

    float tmp_sdata = 0.0f; 
    while (id < n) { 
        tmp_sdata += d_in[id] + d_in[id + blockSize];
        id += gridSize;
    }
    sdata[tid] = tmp_sdata;
    __syncthreads();

    //完全展开
    if (blockSize >= 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
        }
        __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
        }
        __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
        }
        __syncthreads();
    }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}

int main()
{
    const int block_num = 2048;
    const int Num_per_block = N / block_num;
    const int Num_per_thread = Num_per_block / Thread_per_block;

    long int size = N * sizeof(float);
    float* in = (float*)malloc(size);
    float* res = (float*)malloc(block_num * sizeof(float));
    float* d_in;
    float* d_out;

    for (long int i = 0; i < N; i++) {
        in[i] = 1.0;
    }
    for (int i = 0; i < block_num; i++) {
        float cur = 0;
        for (int j = 0; j < Num_per_block; j++) {
            cur += in[i * Num_per_block + j];
        }
        res[i] = cur;
    }

    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, block_num * sizeof(float));
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

    dim3 gridsize(block_num);
    dim3 blocksize(Thread_per_block);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < 1000; i++) {
        reduce6<Thread_per_block> << <gridsize, blocksize >> > (d_in, d_out,N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float   elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    float* out = (float*)malloc(block_num * sizeof(float));
    hipMemcpy(out, d_out, block_num * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if (check(out, res, block_num))printf("the ans is right\n");
    else {
        printf("the ans is wrong\n");
        for (int i = 0; i < block_num; i++) {
            printf("%lf ", out[i]);
        }
        printf("\n");
    }
    cout << "Time is " << elapsedTime << " ms " << endl;
    hipFree(d_in);
    hipFree(d_out);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(in);


    return 0;
}
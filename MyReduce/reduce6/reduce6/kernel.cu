#include "hip/hip_runtime.h"
﻿#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>

#define THREAD_PER_BLOCK 256

template <unsigned int blockSize>
__device__ void warpReduce(volatile float* cache, unsigned int tid) {
	if (blockSize >= 64)cache[tid] += cache[tid + 32];
	if (blockSize >= 32)cache[tid] += cache[tid + 16];
	if (blockSize >= 16)cache[tid] += cache[tid + 8];
	if (blockSize >= 8)cache[tid] += cache[tid + 4];
	if (blockSize >= 4)cache[tid] += cache[tid + 2];
	if (blockSize >= 2)cache[tid] += cache[tid + 1];
}

template <unsigned int blockSize, int NUM_PER_THREAD>
__global__ void reduce6(float* d_in, float* d_out, unsigned int n) {
	__shared__ float sdata[blockSize];

	// each thread loads NUM_PER_THREAD element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockSize * NUM_PER_THREAD) + threadIdx.x;

	sdata[tid] = 0;

#pragma unroll
	for (int iter = 0; iter < NUM_PER_THREAD; iter++) {
		sdata[tid] += d_in[i + iter * blockSize];
	}

	__syncthreads();

	// do reduction in shared mem
	if (blockSize >= 512) {
		if (tid < 256) {
			sdata[tid] += sdata[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			sdata[tid] += sdata[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			sdata[tid] += sdata[tid + 64];
		}
		__syncthreads();
	}
	if (tid < 32) warpReduce<blockSize>(sdata, tid);

	// write result for this block to global mem
	if (tid == 0) d_out[blockIdx.x] = sdata[0];
}


bool check(float* out, float* res, int n) {
	for (int i = 0; i < n; i++) {
		if (out[i] != res[i])
			return false;
	}
	return true;
}

int main() {
	const int N = 32 * 1024 * 1024;
	float* a = (float*)malloc(N * sizeof(float));
	float* d_a;
	hipMalloc((void**)&d_a, N * sizeof(float));

	const int block_num = 1024;
	const int NUM_PER_BLOCK = N / block_num;
	const int NUM_PER_THREAD = NUM_PER_BLOCK / THREAD_PER_BLOCK;
	float* out = (float*)malloc(block_num * sizeof(float));
	float* d_out;
	hipMalloc((void**)&d_out, block_num * sizeof(float));
	float* res = (float*)malloc(block_num * sizeof(float));

	for (int i = 0; i < N; i++) {
		a[i] = i % 456;
	}

	for (int i = 0; i < block_num; i++) {
		float cur = 0;
		for (int j = 0; j < NUM_PER_BLOCK; j++) {
			if (i * NUM_PER_BLOCK + j < N) {
				cur += a[i * NUM_PER_BLOCK + j];
			}
		}
		res[i] = cur;
	}

	hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

	dim3 Grid(block_num, 1);
	dim3 Block(THREAD_PER_BLOCK, 1);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float msecTotal = 0;
	int nIter = 1000;
	hipEventRecord(start);
	for (int i = 0; i < nIter; i++) {
		reduce6<THREAD_PER_BLOCK, NUM_PER_THREAD> << <Grid, Block >> > (d_a, d_out, N);

	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&msecTotal, start, stop);

	hipMemcpy(out, d_out, block_num * sizeof(float), hipMemcpyDeviceToHost);

	if (check(out, res, block_num))printf("the ans is right, time : %f\n", msecTotal);
	else {
		printf("the ans is wrong\n");
		for (int i = 0; i < block_num; i++) {
			printf("%lf ", out[i]);
		}
		printf("\n");
	}

	hipFree(d_a);
	hipFree(d_out);
}
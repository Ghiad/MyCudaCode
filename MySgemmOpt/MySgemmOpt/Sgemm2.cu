#include "hip/hip_runtime.h"
#include "Sgemm2.cuh"

//利用shared memory解决global memory重复读取的问题，每个线程解决一个元素的计算
template <
	const int BLOCK_SIZE_M,
	const int BLOCK_SIZE_N,
	const int BLOCK_SIZE_K
>
__global__ void Sgemm2(float* A, float* B, float* C, const int M, const int N, const int K) {
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int x = by * blockDim.y + ty;
	int y = bx * blockDim.x + tx;
	//对A进行转置
	__shared__ float As[BLOCK_SIZE_K][BLOCK_SIZE_M];
	__shared__ float Bs[BLOCK_SIZE_K][BLOCK_SIZE_N];
	float accum = 0.f;
	for (int i = 0; i < K; i += BLOCK_SIZE_K) {
		//一个线程负责读一个数据,相邻的线程读取相邻的元素,有利于合并访存
		As[tx][ty] = A[x * K + tx + i];//按行读按列存
		Bs[ty][tx] = B[(ty + i) * K + y];//按行读按行存
		__syncthreads();

		for (int j = 0; j < BLOCK_SIZE_M; j++) {
			accum += As[j][ty] * Bs[j][tx];
		}
		__syncthreads;
	}
	C[x * K + y] = accum;
}

void invokSgemm2() {
	const int m = 2048;
	const int n = 2048;
	const int k = 2048;

	size_t bytes_A = m * k * sizeof(float);
	size_t bytes_B = k * n * sizeof(float);
	size_t bytes_C = m * n * sizeof(float);
	float* A = (float*)malloc(bytes_A);
	float* B = (float*)malloc(bytes_B);
	float* C = (float*)malloc(bytes_C);
	float* C1 = (float*)malloc(bytes_C);

	float* dA, * dB, * dC;
	hipMalloc((void**)&dA, bytes_A);
	hipMalloc((void**)&dB, bytes_B);
	hipMalloc((void**)&dC, bytes_C);

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < k; j++) {
			A[i * k + j] = 1.0;
		}
	}
	for (int i = 0; i < k; i++) {
		for (int j = 0; j < n; j++) {
			B[i * k + j] = 1.0;
		}
	}
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			C[i * k + j] = -1.0;
		}
	}

	//测量参数配置
	double msecPerMatrixMul[2] = { 0, 0 };
	double gigaFlops[2] = { 0, 0 };
	double flopsPerMatrixMul = 2.0 * m * n * k;

	//核函数参数配置
	const int BLOCK_SIZE_M = 16;
	const int BLOCK_SIZE_N = 16;
	const int BLOCK_SIZE_K = 16;
	const int THREAD_SIZE_Y = 1;
	const int THREAD_SIZE_X = 1;
	//dim3 gridsize(m / 128, n / 128) 不能写死啊
	dim3 gridsize(m / BLOCK_SIZE_M, n / BLOCK_SIZE_N);
	dim3 blocksize(BLOCK_SIZE_M / THREAD_SIZE_Y, BLOCK_SIZE_K / THREAD_SIZE_X);

	//拷贝数据到device
	hipMemcpy(dA, A, bytes_A, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, bytes_B, hipMemcpyHostToDevice);
	hipMemcpy(dC, C, bytes_C, hipMemcpyHostToDevice);

	//运行核函数
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float msecTotal = 0;
	int nIter = 100;
	hipEventRecord(start);
	for (int run = 0; run < nIter; run++) {
		Sgemm2<BLOCK_SIZE_M, BLOCK_SIZE_N, BLOCK_SIZE_K> << <gridsize, blocksize >> > (dA, dB, dC, m, n, k);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&msecTotal, start, stop);

	hipMemcpy(C, dC, bytes_C, hipMemcpyDeviceToHost);

	msecPerMatrixMul[0] = msecTotal / nIter;
	gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
	printf("My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
		gigaFlops[0],
		msecPerMatrixMul[0],
		flopsPerMatrixMul);

	//运行cublas
	hipblasHandle_t blas_handle;
	hipblasCreate(&blas_handle);
	float alpha = 1.0;
	float beta = 0;
	hipMemcpy(dC, C, bytes_C, hipMemcpyHostToDevice);
	hipEventRecord(start);
	for (int run = 0; run < nIter; run++) {
		hipblasSgemm(blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
			m, n, k, &alpha,
			dA, k, dB, n, &beta, dC, m
		);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&msecTotal, start, stop);
	hipMemcpy(C1, dC, bytes_C, hipMemcpyDeviceToHost);

	msecPerMatrixMul[1] = msecTotal / nIter;
	gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
	printf("CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
		gigaFlops[1],
		msecPerMatrixMul[1],
		flopsPerMatrixMul);
	hipblasDestroy(blas_handle);

	//检测运行结果的正确性
	double eps = 1.e-6;  // machine zero
	bool correct = true;
	for (int i = 0; i < m * n; i++) {
		int row = i / n;
		int col = i % n;
		double abs_err = fabs(C[i] - C1[col * m + row]);
		double dot_length = m;
		double abs_val = fabs(C[i]);
		double rel_err = abs_err / abs_val / dot_length;
		if (rel_err > eps) {
			printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
				i, C[i], C1[col * m + row], eps);
			correct = false;
			break;
		}
	}
	printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
	printf("ratio= %f%%\n", gigaFlops[0] / gigaFlops[1]*100);

	//释放内存
	free(A);
	free(B);
	free(C);
	free(C1);
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
}
#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include<iostream>
#include <malloc.h>
using namespace std;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

template <
    const int BLOCK_SIZE_M,
    const int BLOCK_SIZE_N,
    const int BLOCK_SIZE_K,
    const int THREAD_SIZE_Y,
    const int THREAD_SIZE_X
>
__global__ void Sgemm(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C,const int M,const int N,const int K){
    //参数准备
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    
    //针对C分配线程
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;
    
    //int tid = tx + ty * blockDim.x;和下面效果应该相等
    int tid = tx + ty * THREAD_X_PER_BLOCK;

    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int A_TILE_ROW = tid / A_TILE_THREAD_PER_ROW;
    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4;
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;

    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;
    const int B_TILE_ROW = tid / B_TILE_THREAD_PER_ROW;
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;
    // 每个线程一次取4个float,需要取多少次
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (4 * THREAD_NUM_PER_BLOCK);
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (4 * THREAD_NUM_PER_BLOCK);
    //双倍缓存
    __shared__ float As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = {0};
    float frag_A[2][THREAD_SIZE_Y];
    float frag_B[2][THREAD_SIZE_X];

    float ldg_a_reg[4 * ldg_num_a];
    float ldg_b_reg[4 * ldg_num_b];

    //方便后续global memory的读取，我只关注我要读取的global中在这次大迭代中会取到的数据
    A = &A[by * BLOCK_SIZE_M * K];
    B = &B[bx * BLOCK_SIZE_N];

    //预取数据
    //预取A --> As,索引计算
    for (int i = 0; i < BLOCK_SIZE_M; i += A_TILE_ROW_STRIDE) {
        int ldg_index = i / A_TILE_ROW_STRIDE * 4;
        //我们这只是取第一次的数据所以列方面不需要加上bx*bk
        FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(A_TILE_ROW+i, A_TILE_COL, K)]);

        As[0][A_TILE_COL][A_TILE_ROW] = ldg_a_reg[ldg_index];
        As[0][A_TILE_COL +1][A_TILE_ROW] = ldg_a_reg[ldg_index+1];
        As[0][A_TILE_COL +2][A_TILE_ROW] = ldg_a_reg[ldg_index+2];
        As[0][A_TILE_COL +3][A_TILE_ROW] = ldg_a_reg[ldg_index+3];
    }
    //预取B --> Bs,为啥b不用通过寄存器
    for (int i = 0; i < BLOCK_SIZE_M; i += B_TILE_ROW_STRIDE) {       
        FETCH_FLOAT4(Bs[0][B_TILE_ROW][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(B_TILE_ROW+i,B_TILE_COL,N)]);
    }
    __syncthreads();
    //预取As-->frag_A,考虑扩展性不要写8，而是thread_size_y
    for (int i = 0; i < THREAD_SIZE_Y; i+=4) {
        //FETCH_FLOAT4(frag_A[0][i]) = FETCH_FLOAT4(As[0][0][ty * 8 + i]);
        FETCH_FLOAT4(frag_A[0][i]) = FETCH_FLOAT4(As[0][0][ty * THREAD_SIZE_Y + i]);
    }
    //预取Bs-->frag_B
    for (int i = 0; i < THREAD_SIZE_X; i+=4) {
        FETCH_FLOAT4(frag_B[0][i]) = FETCH_FLOAT4(Bs[0][0][tx * THREAD_SIZE_X + i]);
    }

    int write_stage_idx = 1;
    int load_stage_idx = write_stage_idx ^ 1;
    //开始大迭代
    for (int i = 0; i < K;) {
        i += BLOCK_SIZE_K;
        //如果还有下一次循环，那么就预取数据，从global-->ldg
        if (i < K) {
            for (int j = 0; j < BLOCK_SIZE_M; j += A_TILE_ROW_STRIDE) {
                int ldg_index = j / A_TILE_ROW_STRIDE * 4;
                //大迭代还是在一个block上的概念而不是Block间
                //FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(A_TILE_ROW + j, A_TILE_COL + bx * BLOCK_SIZE_K, K)]);
                FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(A_TILE_ROW + j, A_TILE_COL + i, K)]);
            }
            for (int j = 0; j < BLOCK_SIZE_K; j += B_TILE_ROW_STRIDE) {
                int ldg_index = j / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_b_reg[ldg_index]) = FETCH_FLOAT4(B[OFFSET(B_TILE_ROW + j + i, B_TILE_COL, N)]);
            }
        }
        //开始小循环
        for (int j = 0; j < BLOCK_SIZE_K - 1; j++) {
            for (int k = 0; k < THREAD_SIZE_Y; k+=4) {
                FETCH_FLOAT4(frag_A[(j + 1) % 2][k]) = FETCH_FLOAT4(As[load_stage_idx][j + 1][ty * THREAD_SIZE_Y + k]);
            }
            for (int k = 0; k < THREAD_SIZE_Y; k += 4) {
                FETCH_FLOAT4(frag_B[(j + 1) % 2][k]) = FETCH_FLOAT4(Bs[load_stage_idx][j + 1][tx * THREAD_SIZE_X + k]);
            }
            for (int m = 0; m < THREAD_SIZE_Y; m++) {
                for (int n = 0; n < THREAD_SIZE_X; n++) {
                    //如果上次预取的数据还没取完,会怎么样
                    accum[m][n] += frag_A[j % 2][m] * frag_B[j % 2][n];
                }
            }
        }

        //如果还有下一次循环，预取数据从ldg-->shared
        if (i < K) {
            for (int j = 0; j < BLOCK_SIZE_M; j += A_TILE_ROW_STRIDE) {
                int idx = j / A_TILE_ROW_STRIDE * 4;
                As[write_stage_idx][A_TILE_COL][A_TILE_ROW + j] = ldg_a_reg[idx];
                As[write_stage_idx][A_TILE_COL+1][A_TILE_ROW + j] = ldg_a_reg[idx +1];
                As[write_stage_idx][A_TILE_COL+2][A_TILE_ROW + j] = ldg_a_reg[idx +2];
                As[write_stage_idx][A_TILE_COL+3][A_TILE_ROW + j] = ldg_a_reg[idx +3];
            }
            for (int j = 0; j < BLOCK_SIZE_K; j += B_TILE_ROW_STRIDE) {
                int idx = j / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW + j][B_TILE_COL]) = FETCH_FLOAT4(ldg_b_reg[idx]);
            }
            //对shared memory需要同步
            __syncthreads();
            write_stage_idx ^= 1;
            load_stage_idx = write_stage_idx ^ 1;
        }
        
        //计算最后一次小循环,BLOCK_SIZE_K一般是偶数,所以上面的循环必定会在为frag_A[1]存数据时停止，
        //同时要为下一次大循环预取数据
        for (int k = 0; k < THREAD_SIZE_Y; k += 4) {
            FETCH_FLOAT4(frag_A[0][k]) = FETCH_FLOAT4(As[load_stage_idx][0][ty * THREAD_SIZE_Y + k]);
        }
        for (int k = 0; k < THREAD_SIZE_Y; k += 4) {
            FETCH_FLOAT4(frag_B[0][k]) = FETCH_FLOAT4(Bs[load_stage_idx][0][tx * THREAD_SIZE_X + k]);
        }
        for (int m = 0; m < THREAD_SIZE_Y; m++) {
            for (int n = 0; n < THREAD_SIZE_X; n++) {
                //如果上次预取的数据还没取完,会怎么样
                accum[m][n] += frag_A[1][m] * frag_B[1][n];
            }
        }

    }
    //这里为啥不需要__syncthreads();因为accum是每个线程独有的，不需要一致性
    C = &C[OFFSET(by * BLOCK_SIZE_M, bx * BLOCK_SIZE_K, K)];
    for (int m = 0; m < THREAD_SIZE_Y; m+=4) {
        for (int n = 0; n < THREAD_SIZE_X; n+=4) {
            FETCH_FLOAT4(C[OFFSET(ty * THREAD_SIZE_Y + m, tx * THREAD_SIZE_X + n,N)]) = FETCH_FLOAT4(accum[m][n]);
             
        }
    }
   

}


int main() {
    const int m = 2048;
    const int n = 2048;
    const int k = 2048;
    //数据空间分配
    size_t bytes_A = m * k * sizeof(float);
    size_t bytes_B = k * n * sizeof(float);
    size_t bytes_C = m * n * sizeof(float);
    float* A = (float*)malloc(bytes_A);
    float* B = (float*)malloc(bytes_B);
    float* C = (float*)malloc(bytes_C );
    float* C1 = (float*)malloc(bytes_C);
    float* dA, * dB, * dC;
    checkCudaErrors(hipMalloc((void**)&dA, bytes_A));
    checkCudaErrors(hipMalloc((void**)&dB, bytes_B));
    checkCudaErrors(hipMalloc((void**)&dC, bytes_C ));
    
    //数据初始化
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            A[i*k+j] = 1.0;
        }
    }
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < n; j++) {
            B[i * k + j] = 1.0;
        }
    }
    //测量参数配置
    double msecPerMatrixMul[2] = {0, 0};
    double gigaFlops[2] = { 0, 0 };
    double flopsPerMatrixMul = 2.0 * m * n * k;

    //核函数参数配置
    const int BLOCK_SIZE_M = 128;
    const int BLOCK_SIZE_N = 128;
    const int BLOCK_SIZE_K = 8;
    const int THREAD_SIZE_Y = 8;
    const int THREAD_SIZE_X = 8;
    dim3 gridsize(m / 128, n / 128);
    dim3 blocksize(BLOCK_SIZE_M / THREAD_SIZE_Y, BLOCK_SIZE_N / THREAD_SIZE_X);

    //拷贝数据到device
    checkCudaErrors(hipMemcpy(dA, A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dB, B, bytes_B, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dC, C, bytes_C , hipMemcpyHostToDevice));

    //运行核函数
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 1;
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0; run < nIter; run++) {
        Sgemm <BLOCK_SIZE_M, BLOCK_SIZE_N, BLOCK_SIZE_K, THREAD_SIZE_Y, THREAD_SIZE_X> << <gridsize, blocksize >> > (dA, dB, dC, m, n, k);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    checkCudaErrors(hipMemcpy(C, dC, bytes_C, hipMemcpyDeviceToHost));
    msecPerMatrixMul[0] = msecTotal / nIter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf("My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[0],
        msecPerMatrixMul[0],
        flopsPerMatrixMul);

    //运行cublas
    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipMemcpy(dC, C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0; run < nIter; run++) {
        hipblasSgemm(blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
            m, n, k, &alpha,
            dA, k, dB, n, &beta, dC, n
        );
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
    checkCudaErrors(hipMemcpy(C1, dC, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[1] = msecTotal / nIter;
    gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
    printf("CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[1],
        msecPerMatrixMul[1],
        flopsPerMatrixMul);
    hipblasDestroy(blas_handle);

    //检测运行结果的正确性
    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < m * n; i++) {
        int row = i / n;
        int col = i % n;
        double abs_err = fabs(C[i] - C1[col * m + row]);
        double dot_length = m;
        double abs_val = fabs(C[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                i, C[i], C1[col * m + row], eps);
            correct = false;
            break;
        }
    }
    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);

    //释放内存
    free(A);
    free(B);
    free(C);
    free(C1);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    return 0;
}
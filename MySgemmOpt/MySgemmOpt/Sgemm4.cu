#include "hip/hip_runtime.h"
#include "Sgemm4.cuh"
#define OFFSET(row,col,ld) ((row)*(ld)+(col))
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

template <
	const int BLOCK_SIZE_M,
	const int BLOCK_SIZE_N,
	const int BLOCK_SIZE_K,
	const int THREAD_SIZE_Y,
	const int THREAD_SIZE_X
>
__global__ void Sgemm4(float* A, float* B, float* C, const int M, const int N, const int K) {
	//使用双缓冲,加不加pragma unroll对性能影响非常大
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tid = ty * blockDim.x + tx;

	const int THREAD_PER_BLOCK = (BLOCK_SIZE_M * BLOCK_SIZE_N) / (THREAD_SIZE_X * THREAD_SIZE_Y);
	const int A_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
	const int A_START_ROW = tid / A_THREAD_PER_ROW;
	const int A_START_COL = tid % A_THREAD_PER_ROW * 4;
	const int A_ROW_STRIDE = THREAD_PER_BLOCK / A_THREAD_PER_ROW;

	const int B_THREAD_PER_ROW = BLOCK_SIZE_N / 4;
	const int B_START_ROW = tid / B_THREAD_PER_ROW;
	const int B_START_COL = tid % B_THREAD_PER_ROW * 4;
	const int B_ROW_STRIDE = THREAD_PER_BLOCK / B_THREAD_PER_ROW;

	const int A_LDG_NUM = (BLOCK_SIZE_M * BLOCK_SIZE_K) / (4 * THREAD_PER_BLOCK);
	const int B_LDG_NUM = (BLOCK_SIZE_K * BLOCK_SIZE_N) / (4 * THREAD_PER_BLOCK);

	__shared__ float As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];
	__shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
	float frag_A[2][THREAD_SIZE_Y];
	float frag_B[2][THREAD_SIZE_X];
	float ldg_A[4 * A_LDG_NUM];
	float ldg_B[4 * B_LDG_NUM];
	float accum[THREAD_SIZE_Y][THREAD_SIZE_X] = { 0.0 };
	A = &A[by * BLOCK_SIZE_M * K];
	B = &B[bx * BLOCK_SIZE_N];

	//预读取
#pragma unroll
	for (int i = 0; i < BLOCK_SIZE_M; i += A_ROW_STRIDE) {
		int ldg_index = i / A_ROW_STRIDE * 4;
		FETCH_FLOAT4(ldg_A[ldg_index]) = FETCH_FLOAT4(A[OFFSET(A_START_ROW + i,
			A_START_COL,
			K)]);
		As[0][A_START_COL][A_START_ROW + i] = ldg_A[ldg_index];
		As[0][A_START_COL + 1][A_START_ROW + i] = ldg_A[ldg_index + 1];
		As[0][A_START_COL + 2][A_START_ROW + i] = ldg_A[ldg_index + 2];
		As[0][A_START_COL + 3][A_START_ROW + i] = ldg_A[ldg_index + 3];
	}
#pragma unroll
	for (int i = 0; i < BLOCK_SIZE_K; i += B_ROW_STRIDE) {

		FETCH_FLOAT4(Bs[0][B_START_ROW + i][B_START_COL]) = FETCH_FLOAT4(B[OFFSET(B_START_ROW + i,
			B_START_COL,
			N)]);
	}
	__syncthreads();
#pragma unroll
	for (int i = 0; i < THREAD_SIZE_Y; i += 4) {
		FETCH_FLOAT4(frag_A[0][i]) = FETCH_FLOAT4(As[0][0][ty * THREAD_SIZE_Y + i]);
	}
#pragma unroll
	for (int i = 0; i < THREAD_SIZE_X; i += 4) {
		FETCH_FLOAT4(frag_B[0][i]) = FETCH_FLOAT4(Bs[0][0][tx * THREAD_SIZE_X + i]);
	}
	//大迭代开始
	int write_flag = 1;
#pragma unroll
	for (int i = 0; i < K; ) {
		i += BLOCK_SIZE_K;

		//取下一次迭代的数据到ldg中
		if (i < K) {
#pragma unroll
			for (int j = 0; j < BLOCK_SIZE_M; j += A_ROW_STRIDE) {
				int ldg_index = j / A_ROW_STRIDE * 4;
				FETCH_FLOAT4(ldg_A[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
					A_START_ROW + j,
					A_START_COL + i,
					K)]);
			}
#pragma unroll
			for (int j = 0; j < BLOCK_SIZE_K; j += B_ROW_STRIDE) {
				int ldg_index = j / B_ROW_STRIDE * 4;
				FETCH_FLOAT4(ldg_B[ldg_index]) = FETCH_FLOAT4(B[OFFSET(
					B_START_ROW + j + i,
					B_START_COL,
					N)]);
			}
		}


		int load_flag = write_flag ^ 1;
		//用上一次预加载的数据计算，小迭代开始
#pragma unroll
		for (int j = 0; j < BLOCK_SIZE_K - 1; j++) {
			//加载下一次小迭代要用的数据
#pragma unroll
			for (int k = 0; k < THREAD_SIZE_Y; k += 4) {
				FETCH_FLOAT4(frag_A[(j + 1) % 2][k]) = FETCH_FLOAT4(As[load_flag][j + 1][ty * THREAD_SIZE_Y + k]);
			}
#pragma unroll
			for (int k = 0; k < THREAD_SIZE_X; k += 4) {
				FETCH_FLOAT4(frag_B[(j + 1) % 2][k]) = FETCH_FLOAT4(Bs[load_flag][j + 1][tx * THREAD_SIZE_X + k]);
			}
#pragma unroll
			for (int m = 0; m < THREAD_SIZE_Y; m++) {
#pragma unroll
				for (int n = 0; n < THREAD_SIZE_X; n++) {
					accum[m][n] += frag_A[j % 2][m] * frag_B[j % 2][n];
				}
			}
		}

		if (i < K) {
			//从暂存的lag中读取下一次迭代的数据到shared
#pragma unroll
			for (int j = 0; j < BLOCK_SIZE_M; j += A_ROW_STRIDE) {
				int ldg_idx = j / A_ROW_STRIDE * 4;
				As[write_flag][A_START_COL][A_START_ROW + j] = ldg_A[ldg_idx];
				As[write_flag][A_START_COL + 1][A_START_ROW + j] = ldg_A[ldg_idx + 1];
				As[write_flag][A_START_COL + 2][A_START_ROW + j] = ldg_A[ldg_idx + 2];
				As[write_flag][A_START_COL + 3][A_START_ROW + j] = ldg_A[ldg_idx + 3];

			}
#pragma unroll
			for (int j = 0; j < BLOCK_SIZE_K; j += B_ROW_STRIDE) {
				int ldg_idx = j / B_ROW_STRIDE * 4;
				FETCH_FLOAT4(Bs[write_flag][B_START_ROW + j][B_START_COL]) = FETCH_FLOAT4(ldg_B[ldg_idx]);

			}
			__syncthreads();
			write_flag ^= 1;
		}


		//计算最后一个小迭代	
#pragma unroll
		for (int j = 0; j < THREAD_SIZE_Y; j += 4) {
			FETCH_FLOAT4(frag_A[0][j]) = FETCH_FLOAT4(As[load_flag ^ 1][0][ty * THREAD_SIZE_Y + j]);
		}
#pragma unroll
		for (int j = 0; j < THREAD_SIZE_X; j += 4) {
			FETCH_FLOAT4(frag_B[0][j]) = FETCH_FLOAT4(Bs[load_flag ^ 1][0][tx * THREAD_SIZE_X + j]);
		}
#pragma unroll
		for (int m = 0; m < THREAD_SIZE_Y; m++) {
#pragma unroll
			for (int n = 0; n < THREAD_SIZE_X; n++) {
				accum[m][n] += frag_A[1][m] * frag_B[1][n];
			}
		}
	}
#pragma unroll
	for (int m = 0; m < THREAD_SIZE_Y; m++) {
#pragma unroll
		for (int n = 0; n < THREAD_SIZE_X; n += 4) {
			FETCH_FLOAT4(C[OFFSET(
				by * BLOCK_SIZE_M + ty * THREAD_SIZE_Y + m,
				bx * BLOCK_SIZE_N + tx * THREAD_SIZE_X + n,
				N)]) = FETCH_FLOAT4(accum[m][n]);
		}
	}
}

void invokSgemm4() {
	const int m = 2048;
	const int n = 2048;
	const int k = 2048;

	size_t bytes_A = m * k * sizeof(float);
	size_t bytes_B = k * n * sizeof(float);
	size_t bytes_C = m * n * sizeof(float);
	float* A = (float*)malloc(bytes_A);
	float* B = (float*)malloc(bytes_B);
	float* C = (float*)malloc(bytes_C);
	float* C1 = (float*)malloc(bytes_C);

	float* dA, * dB, * dC;
	hipMalloc((void**)&dA, bytes_A);
	hipMalloc((void**)&dB, bytes_B);
	hipMalloc((void**)&dC, bytes_C);

	for (int i = 0; i < m; i++) {
		for (int j = 0; j < k; j++) {
			A[i * k + j] = 1.0;
		}
	}
	for (int i = 0; i < k; i++) {
		for (int j = 0; j < n; j++) {
			B[i * k + j] = 1.0;
		}
	}
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			C[i * k + j] = -1.0;
		}
	}

	//测量参数配置
	double msecPerMatrixMul[2] = { 0, 0 };
	double gigaFlops[2] = { 0, 0 };
	double flopsPerMatrixMul = 2.0 * m * n * k;

	//核函数参数配置
	const int BLOCK_SIZE_M = 128;
	const int BLOCK_SIZE_N = 128;
	const int BLOCK_SIZE_K = 8;
	const int THREAD_SIZE_Y = 8;
	const int THREAD_SIZE_X = 8;
	//dim3 gridsize(m / 128, n / 128) 不能写死啊
	dim3 gridsize(m / BLOCK_SIZE_M, n / BLOCK_SIZE_N);
	dim3 blocksize(BLOCK_SIZE_N / THREAD_SIZE_Y, BLOCK_SIZE_M / THREAD_SIZE_X);

	//拷贝数据到device
	hipMemcpy(dA, A, bytes_A, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, bytes_B, hipMemcpyHostToDevice);
	hipMemcpy(dC, C, bytes_C, hipMemcpyHostToDevice);

	//运行核函数
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float msecTotal = 0;
	int nIter = 1;
	hipEventRecord(start);
	for (int run = 0; run < nIter; run++) {
		Sgemm4<BLOCK_SIZE_M, BLOCK_SIZE_N, BLOCK_SIZE_K, THREAD_SIZE_Y, THREAD_SIZE_X> << <gridsize, blocksize >> > (dA, dB, dC, m, n, k);

	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&msecTotal, start, stop);

	hipMemcpy(C, dC, bytes_C, hipMemcpyDeviceToHost);

	msecPerMatrixMul[0] = msecTotal / nIter;
	gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
	printf("My gemm4 Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
		gigaFlops[0],
		msecPerMatrixMul[0],
		flopsPerMatrixMul);

	//运行cublas
	hipblasHandle_t blas_handle;
	hipblasCreate(&blas_handle);
	float alpha = 1.0;
	float beta = 0;
	hipMemcpy(dC, C, bytes_C, hipMemcpyHostToDevice);
	hipEventRecord(start);
	for (int run = 0; run < nIter; run++) {
		hipblasSgemm(blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
			m, n, k, &alpha,
			dA, k, dB, n, &beta, dC, m
		);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&msecTotal, start, stop);
	hipMemcpy(C1, dC, bytes_C, hipMemcpyDeviceToHost);

	msecPerMatrixMul[1] = msecTotal / nIter;
	gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
	printf("CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
		gigaFlops[1],
		msecPerMatrixMul[1],
		flopsPerMatrixMul);
	hipblasDestroy(blas_handle);

	//检测运行结果的正确性
	double eps = 1.e-6;  // machine zero
	bool correct = true;
	for (int i = 0; i < m * n; i++) {
		int row = i / n;
		int col = i % n;
		double abs_err = fabs(C[i] - C1[col * m + row]);
		double dot_length = m;
		double abs_val = fabs(C[i]);
		double rel_err = abs_err / abs_val / dot_length;
		if (rel_err > eps) {
			printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
				i, C[i], C1[col * m + row], eps);
			correct = false;
			break;
		}
	}
	printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
	printf("ratio= %f%%\n", gigaFlops[0] / gigaFlops[1] * 100);

	//释放内存
	free(A);
	free(B);
	free(C);
	free(C1);
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
}
#include "hip/hip_runtime.h"
﻿// optimize sgemm

#include <stdio.h>
#include <stdlib.h>
#include "assert.h" 

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// cal offset from row col and ld , in row-major matrix, ld is the width of the matrix
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

// transfer float4
#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

// K: ldA
// N: ldB
template <
    const int BLOCK_SIZE_M,  // height of block of C that each thread block calculate
    const int BLOCK_SIZE_K,  // width of block of A that each thread block load into shared memory
    const int BLOCK_SIZE_N,  // width of block of C that each thread block calculate
    const int THREAD_SIZE_Y, // height of block of C that each thread calculate
    const int THREAD_SIZE_X,  // width of block of C that each thread calculate
    const bool ENABLE_DOUBLE_BUFFER // whether enable double buffering or not
>
__global__ void Sgemm(
    float* __restrict__ A,
    float* __restrict__ B,
    float* __restrict__ C,
    const int M,
    const int N,
    const int K) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // the threads number in Block of X,Y
    const int THREAD_X_PER_BLOCK = BLOCK_SIZE_N / THREAD_SIZE_X;
    const int THREAD_Y_PER_BLOCK = BLOCK_SIZE_M / THREAD_SIZE_Y;
    const int THREAD_NUM_PER_BLOCK = THREAD_X_PER_BLOCK * THREAD_Y_PER_BLOCK;

    // thread id in cur Block
    const int tid = ty * THREAD_X_PER_BLOCK + tx;

    // shared memory
    __shared__ float As[2][BLOCK_SIZE_K][BLOCK_SIZE_M];
    __shared__ float Bs[2][BLOCK_SIZE_K][BLOCK_SIZE_N];
    // registers for C
    float accum[THREAD_SIZE_Y][THREAD_SIZE_X];
#pragma unroll
    for (int i = 0; i < THREAD_SIZE_Y; i++) {
#pragma unroll
        for (int j = 0; j < THREAD_SIZE_X; j++) {
            accum[i][j] = 0.0;
        }
    }
    // registers for A and B
    float frag_a[2][THREAD_SIZE_Y];
    float frag_b[2][THREAD_SIZE_X];
    // registers load global memory
    const int ldg_num_a = BLOCK_SIZE_M * BLOCK_SIZE_K / (THREAD_NUM_PER_BLOCK * 4);
    const int ldg_num_b = BLOCK_SIZE_K * BLOCK_SIZE_N / (THREAD_NUM_PER_BLOCK * 4);
    float ldg_a_reg[4 * ldg_num_a];
    float ldg_b_reg[4 * ldg_num_b];

    // threads number in one row
    const int A_TILE_THREAD_PER_ROW = BLOCK_SIZE_K / 4;
    const int B_TILE_THREAD_PER_ROW = BLOCK_SIZE_N / 4;

    // row number and col number that needs to be loaded by this thread
    const int A_TILE_ROW_START = tid / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_START = tid / B_TILE_THREAD_PER_ROW;

    const int A_TILE_COL = tid % A_TILE_THREAD_PER_ROW * 4;
    const int B_TILE_COL = tid % B_TILE_THREAD_PER_ROW * 4;

    // row stride that thread uses to load multiple rows of a tile
    const int A_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / A_TILE_THREAD_PER_ROW;
    const int B_TILE_ROW_STRIDE = THREAD_NUM_PER_BLOCK / B_TILE_THREAD_PER_ROW;

    A = &A[(BLOCK_SIZE_M * by) * K];
    B = &B[BLOCK_SIZE_N * bx];

    //load index of the tile
    const int warp_id = tid / 32;
    const int lane_id = tid % 32;
    const int a_tile_index = warp_id / 2 * 16 + lane_id / 8 * 4; //warp_id * 8 + (lane_id / 16)*4; // (warp_id/4)*32 + ((lane_id%16)/2)*4;
    const int b_tile_index = warp_id % 2 * 32 + lane_id % 8 * 4; //(lane_id % 16) * 4; // (warp_id%4)*16 + (lane_id/16)*8 + (lane_id%2)*4;

    //transfer first tile from global mem to shared mem
    // load A from global memory to shared memory
#pragma unroll
    for (int i = 0; i < BLOCK_SIZE_M; i += A_TILE_ROW_STRIDE) {
        int ldg_index = i / A_TILE_ROW_STRIDE * 4;
        FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
            A_TILE_ROW_START + i, // row
            A_TILE_COL, // col
            K)]);
        As[0][A_TILE_COL][A_TILE_ROW_START + i] = ldg_a_reg[ldg_index];
        As[0][A_TILE_COL + 1][A_TILE_ROW_START + i] = ldg_a_reg[ldg_index + 1];
        As[0][A_TILE_COL + 2][A_TILE_ROW_START + i] = ldg_a_reg[ldg_index + 2];
        As[0][A_TILE_COL + 3][A_TILE_ROW_START + i] = ldg_a_reg[ldg_index + 3];
    }
    // load B from global memory to shared memory
#pragma unroll
    for (int i = 0; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
        FETCH_FLOAT4(Bs[0][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(B[OFFSET(
            B_TILE_ROW_START + i, // row
            B_TILE_COL, // col
            N)]);
    }
    __syncthreads();

    // load A from shared memory to register
    FETCH_FLOAT4(frag_a[0][0]) = FETCH_FLOAT4(As[0][0][a_tile_index]);
    FETCH_FLOAT4(frag_a[0][4]) = FETCH_FLOAT4(As[0][0][a_tile_index + 64]);

    // load B from shared memory to register
    FETCH_FLOAT4(frag_b[0][0]) = FETCH_FLOAT4(Bs[0][0][b_tile_index]);
    FETCH_FLOAT4(frag_b[0][4]) = FETCH_FLOAT4(Bs[0][0][b_tile_index + 64]);

    int write_stage_idx = 1;
    int tile_idx = 0;
    do {
        // next tile index
        tile_idx += BLOCK_SIZE_K;
        // load next tile from global mem
        if (tile_idx < K) {
#pragma unroll
            for (int i = 0; i < BLOCK_SIZE_M; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_a_reg[ldg_index]) = FETCH_FLOAT4(A[OFFSET(
                    A_TILE_ROW_START + i, // row
                    A_TILE_COL + tile_idx, // col
                    K)]);
            }
#pragma unroll
            for (int i = 0; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(ldg_b_reg[ldg_index]) = FETCH_FLOAT4(B[OFFSET(
                    tile_idx + B_TILE_ROW_START + i, // row
                    B_TILE_COL, // col
                    N)]);
            }
        }

        int load_stage_idx = write_stage_idx ^ 1;

#pragma unroll
        for (int j = 0; j < BLOCK_SIZE_K - 1; ++j) {
            // load next tile from shared mem to register 
            // load A from shared memory to register
            FETCH_FLOAT4(frag_a[(j + 1) % 2][0]) = FETCH_FLOAT4(As[load_stage_idx][(j + 1)][a_tile_index]);
            FETCH_FLOAT4(frag_a[(j + 1) % 2][4]) = FETCH_FLOAT4(As[load_stage_idx][(j + 1)][a_tile_index + 64]);
            // load B from shared memory to register
            FETCH_FLOAT4(frag_b[(j + 1) % 2][0]) = FETCH_FLOAT4(Bs[load_stage_idx][(j + 1)][b_tile_index]);
            FETCH_FLOAT4(frag_b[(j + 1) % 2][4]) = FETCH_FLOAT4(Bs[load_stage_idx][(j + 1)][b_tile_index + 64]);
            // compute C THREAD_SIZE_X x THREAD_SIZE_Y
#pragma unroll
            for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
#pragma unroll
                for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                    accum[thread_y][thread_x] += frag_a[j % 2][thread_y] * frag_b[j % 2][thread_x];
                }
            }
        }

        if (tile_idx < K) {
            // load A from global memory to shared memory
#pragma unroll
            for (int i = 0; i < BLOCK_SIZE_M; i += A_TILE_ROW_STRIDE) {
                int ldg_index = i / A_TILE_ROW_STRIDE * 4;
                As[write_stage_idx][A_TILE_COL][A_TILE_ROW_START + i] = ldg_a_reg[ldg_index];
                As[write_stage_idx][A_TILE_COL + 1][A_TILE_ROW_START + i] = ldg_a_reg[ldg_index + 1];
                As[write_stage_idx][A_TILE_COL + 2][A_TILE_ROW_START + i] = ldg_a_reg[ldg_index + 2];
                As[write_stage_idx][A_TILE_COL + 3][A_TILE_ROW_START + i] = ldg_a_reg[ldg_index + 3];
            }
            // load B from global memory to shared memory
#pragma unroll
            for (int i = 0; i < BLOCK_SIZE_K; i += B_TILE_ROW_STRIDE) {
                int ldg_index = i / B_TILE_ROW_STRIDE * 4;
                FETCH_FLOAT4(Bs[write_stage_idx][B_TILE_ROW_START + i][B_TILE_COL]) = FETCH_FLOAT4(ldg_b_reg[ldg_index]);
            }
            // use double buffer, only need one sync
            __syncthreads();
            // switch
            write_stage_idx ^= 1;
        }

        // load first tile from shared mem to register of next iter
        // load A from shared memory to register
        FETCH_FLOAT4(frag_a[0][0]) = FETCH_FLOAT4(As[load_stage_idx ^ 1][0][a_tile_index]);
        FETCH_FLOAT4(frag_a[0][4]) = FETCH_FLOAT4(As[load_stage_idx ^ 1][0][a_tile_index + 64]);
        // load B from shared memory to register
        FETCH_FLOAT4(frag_b[0][0]) = FETCH_FLOAT4(Bs[load_stage_idx ^ 1][0][b_tile_index]);
        FETCH_FLOAT4(frag_b[0][4]) = FETCH_FLOAT4(Bs[load_stage_idx ^ 1][0][b_tile_index + 64]);
        // compute C THREAD_SIZE_X x THREAD_SIZE_Y
#pragma unroll
        for (int thread_y = 0; thread_y < THREAD_SIZE_Y; ++thread_y) {
#pragma unroll
            for (int thread_x = 0; thread_x < THREAD_SIZE_X; ++thread_x) {
                accum[thread_y][thread_x] += frag_a[1][thread_y] * frag_b[1][thread_x];
            }
        }
    } while (tile_idx < K);

    const int c_block_row = a_tile_index;
    const int c_block_col = b_tile_index;

    //store C00 block
    for (int i = 0; i < 4; i++) {
        FETCH_FLOAT4(C[OFFSET(
            BLOCK_SIZE_M * by + c_block_row + i,
            BLOCK_SIZE_N * bx + c_block_col,
            N)]) = FETCH_FLOAT4(accum[i][0]);
    }
    //store C01 block
    for (int i = 0; i < 4; i++) {
        FETCH_FLOAT4(C[OFFSET(
            BLOCK_SIZE_M * by + c_block_row + i,
            BLOCK_SIZE_N * bx + c_block_col + 64,
            N)]) = FETCH_FLOAT4(accum[i][4]);
    }
    //store C10 block
    for (int i = 0; i < 4; i++) {
        FETCH_FLOAT4(C[OFFSET(
            BLOCK_SIZE_M * by + c_block_row + 64 + i,
            BLOCK_SIZE_N * bx + c_block_col,
            N)]) = FETCH_FLOAT4(accum[i + 4][0]);
    }
    //store C11 block
    for (int i = 0; i < 4; i++) {
        FETCH_FLOAT4(C[OFFSET(
            BLOCK_SIZE_M * by + c_block_row + 64 + i,
            BLOCK_SIZE_N * bx + c_block_col + 64,
            N)]) = FETCH_FLOAT4(accum[i + 4][4]);
    }
}

int main() {
    
    size_t M = 1024;
    size_t K = 1024;
    size_t N = 1024;

    assert(M % 8 == 0);
    assert(N % 8 == 0);
    assert(K % 8 == 0);

    size_t bytes_A = sizeof(float) * M * K;
    size_t bytes_B = sizeof(float) * K * N;
    size_t bytes_C = sizeof(float) * M * N;
    float* h_A = (float*)malloc(bytes_A);
    float* h_B = (float*)malloc(bytes_B);
    float* h_C = (float*)malloc(bytes_C);
    float* h_C1 = (float*)malloc(bytes_C);

    float* d_A;
    float* d_B;
    float* d_C;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_B, bytes_B));
    checkCudaErrors(hipMalloc(&d_C, bytes_C));
    double msecPerMatrixMul[2] = { 0, 0 };
    double gigaFlops[2] = { 0, 0 };
    double flopsPerMatrixMul = 2.0 * M * N * K;

    // don't edit it
    const int BLOCK_SIZE_M = 128;
    const int BLOCK_SIZE_K = 8;
    const int BLOCK_SIZE_N = 128;
    const int THREAD_SIZE_X = 8;
    const int THREAD_SIZE_Y = 8;
    const bool ENABLE_DOUBLE_BUFFER = false;

    // 生成A的数据
    for (int i = 0; i < M * K; i++) {
        h_A[i] = i / 13;
    }

    // 生成B的数据
    for (int i = 0; i < K * N; i++) {
        h_B[i] = i % 13;
    }

    checkCudaErrors(hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, bytes_B, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msecTotal = 0;
    int nIter = 1000;

    checkCudaErrors(hipMemcpy(d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0; run < nIter; run++) {
        dim3 dimBlock(BLOCK_SIZE_N / THREAD_SIZE_X, BLOCK_SIZE_M / THREAD_SIZE_Y);
        dim3 dimGrid(N / BLOCK_SIZE_N, M / BLOCK_SIZE_M);
        Sgemm<BLOCK_SIZE_M, BLOCK_SIZE_K, BLOCK_SIZE_N, THREAD_SIZE_Y, THREAD_SIZE_X, ENABLE_DOUBLE_BUFFER>
            << < dimGrid, dimBlock >> > (d_A, d_B, d_C, M, N, K);
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));


    checkCudaErrors(hipMemcpy(h_C, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[0] = msecTotal / nIter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf("My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[0],
        msecPerMatrixMul[0],
        flopsPerMatrixMul);

    // cublas

    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    float alpha = 1.0;
    float beta = 0;
    checkCudaErrors(hipMemcpy(d_C, h_C, bytes_C, hipMemcpyHostToDevice));
    checkCudaErrors(hipEventRecord(start));
    for (int run = 0; run < nIter; run++) {
        hipblasSgemm(blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
            M, N, K, &alpha,
            d_A, K, d_B, N, &beta, d_C, N
        );
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    checkCudaErrors(hipMemcpy(h_C1, d_C, bytes_C, hipMemcpyDeviceToHost));

    msecPerMatrixMul[1] = msecTotal / nIter;
    gigaFlops[1] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
    printf("CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        gigaFlops[1],
        msecPerMatrixMul[1],
        flopsPerMatrixMul);

    hipblasDestroy(blas_handle);


    double eps = 1.e-6;  // machine zero
    bool correct = true;
    for (int i = 0; i < M * N; i++) {
        int row = i / N;
        int col = i % N;
        double abs_err = fabs(h_C[i] - h_C1[col * M + row]);
        double dot_length = M;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps) {
            printf("Error! Matrix[%d][%d]=%.8f, ref=%.8f error term is > %E\n",
                row, col, h_C[i], h_C1[col * M + row], eps);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);

    // Free Memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C1);
}
